#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <unistd.h>
#include <stdio.h>

#include ""
#include "hip/hip_runtime.h"
#include "utils.h"

__device__ int binarySearch(int* arr, int val, int low, int high) {
	if (high <= low) 
        return (val > arr[low]) ? (low + 1) : low; 
  
    int mid = (low + high)/2; 
  
    //if(val == a[mid]) we dont support duplicates anyway
        //return mid+1; 
  
    if(val > arr[mid])
		return binarySearch(arr, val, mid+1, high);
		
    return binarySearch(arr, val, low, mid); // was mid-1
}

// Should use binary search!
__device__ int getIndex(int* subAux, int ownIndex, int nLow, int nTot) {
	//int numBeforeInOther = 0;
	//int indexInArr = 0;
	int scanIndex;
	int upperBound;
	bool partOfFirstArr = ownIndex < nLow;

	if(partOfFirstArr) {
		scanIndex = nLow; // Start scanning in 2nd arr
		upperBound = nTot;
	} 
	else {
		scanIndex = 0;
		upperBound = nLow;
	}

	//while (subAux[scanIndex] < subAux[ownIndex] && scanIndex < upperBound)
		//scanIndex++;

	//scanIndex++;

	//numBeforeInOther = partOfFirstArr ? scanIndex - nLow : scanIndex;

	scanIndex = binarySearch(subAux, subAux[ownIndex], scanIndex, upperBound-1);
	// printf("ScanIndex found: %d, by thread: %d, nTot: %d\n", scanIndex, ownIndex, nTot);
	// numBeforeInOther = partOfFirstArr ? indexInArr - nLow : indexInArr;

	return ownIndex + scanIndex - nLow;

	/*
	if(! partOfFirstArr)
		return ownIndex - nLow + numBeforeInOther;

	return ownIndex + numBeforeInOther;
	*/
}
// CANNOT HANDLE DUPLICATES
__global__ void mergeKernel(int* arr, int* aux, int low, int mid, int high) {
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int nLow = mid - low + 1; // optimize
	int nHigh = high - mid;
	int nTot = nLow + nHigh;
	
	if(idx >= nTot)
		return;

	int arrIndex = getIndex(&aux[low], idx, nLow, nTot);
	arr[low + arrIndex] = aux[low + idx];
	
	//printf("idx %d assigns %d to %d\n", idx, aux[low + idx], low + arrIndex);
}

// Just a sequential merge instead of nested kernel
__device__ void merge(int* arr, int* aux, int low, int mid, int high) {
	int i = 0;
	int j = 0;
	int mergedIndex = low;

	int nLow = mid - low + 1;
	int nHigh = high - mid;

	while (i < nLow && j < nHigh) {
		if (aux[low + i] <= aux[mid + 1 + j]) {
			arr[mergedIndex] = aux[low + i];
			i++;
		}
		else {
			arr[mergedIndex] = aux[mid + 1 + j];
			j++;
		}
		mergedIndex++;
	}

	while (i < nLow) {
		arr[mergedIndex] = aux[low + i];
		i++;
		mergedIndex++;
	}
	while (j < nHigh) {
		arr[mergedIndex] = aux[mid + 1 + j];
		j++;
		mergedIndex++;
	}
}

__global__ void mergeSort(int* arr, int* aux, int currentSize, int n, int width) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int low = idx * width;
	if(low >= n) return;
	int mid = low + currentSize - 1;
	int high = min(low + width - 1, n-1);

	int nTot = high - low + 1; // number of threads to spawn
	if(nTot > 4096) { // Don't launch a kernel if the merge is small
		int numThreadsPerBlock = 256;
		int numBlocks = (nTot + numThreadsPerBlock - 1) / numThreadsPerBlock;
		//printf("ID %d: Spawning %d threads to merge %d elements\n", idx, numThreadsPerBlock*numBlocks, nTot);
		// merge(arr, aux, low, mid, high);
		mergeKernel<<<numBlocks, numThreadsPerBlock>>>(arr, aux, low, mid, high);
		//__syncthreads();
	} else {
		merge(arr, aux, low, mid, high);
	}

}

void mergeSortGPU(int* arr, int n) { // ASSUMES POWER OF 2 FOR NOW

	int* deviceArr;
	int* auxArr;

	hipMallocManaged(&deviceArr, n * sizeof(int));
	hipMallocManaged(&auxArr, n * sizeof(int)); // Allocate aux arr on GPU
	hipMemcpy(deviceArr, arr, n * sizeof(int), hipMemcpyDefault); // Move arr to cuda managed memory

	for (int currentSize = 1; currentSize < n; currentSize *= 2) {

		int width = currentSize*2;
		int numSorts = n / width; // number of sorting threads to spawn

		int numThreadsPerBlock = 256;
		int numBlocks = (numSorts + numThreadsPerBlock - 1) / numThreadsPerBlock;
		
		
		hipMemcpy(auxArr, deviceArr, n * sizeof(int), hipMemcpyDeviceToDevice); 
		mergeSort<<<numBlocks, numThreadsPerBlock>>>(deviceArr, auxArr, currentSize, n, width);
		
		// std::cout << "Num sorts: " << numSorts << ", num threads spawned: " << numThreadsPerBlock * numBlocks << std::endl;
		// usleep(1000000);
		// hipMemcpy(arr, deviceArr, n * sizeof(int), hipMemcpyDefault);
		// std::cout << "DevArr transferred to arr: ";
		// printArray(arr, n);
	}

	hipMemcpy(arr, deviceArr, n * sizeof(int), hipMemcpyDefault);

	hipFree(deviceArr);
	hipFree(auxArr);
}